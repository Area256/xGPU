#include "hip/hip_runtime.h"
/*

  Simple cross-product, outputs in correct triangular form.

  - Coalescing memory access in all reads
  - No memory coalscing in writes (will be fixed)
  - Shared memory reads of type float2 to reduce global memory traffic
  - Each thread works on a 2x2 tile of data

  On a GTX 480 with >= 512 tiles this kernel achieve in excess of a
  teraflop.

 */

#include <stdio.h>

#include "xgpu.h"
#include "xgpu_info.h"
#include "xgpu_version.h"
#include "cube/cube.h"

// Set data types accordingly
#ifndef FIXED_POINT
#define COMPLEX_INPUT float2
#define SCALE 1.0f // no rescale required for FP32
#else
#define COMPLEX_INPUT char2 
#define SCALE 16129.0f // need to rescale result 
#endif // FIXED_POINT

// whether we are writing the matrix back to device memory (used for benchmarking)
static int writeMatrix = 1;
// this must be enabled for this option to work though, slightly hurts performance
//#define WRITE_OPTION 

typedef struct XGPUInternalContextStruct {
  //memory pointers on the device
  ComplexInput *array_d[2];
  Complex *matrix_d;

  // used for overlapping comms and compute
  hipStream_t *streams;

  // texture channel descriptor
  hipChannelFormatDesc channelDesc;

  // Host input array that we allocated and should free
  ComplexInput * free_array_h;

  // Host input array that we registered and should unregister
  ComplexInput * unregister_array_h;

  // Host output array that we allocated and should free
  Complex * free_matrix_h;

  // Host output array that we registered and should unregister
  Complex * unregister_matrix_h;
} XGPUInternalContext;

#define TILE_HEIGHT 8
#define TILE_WIDTH 8
#define NPOL 2

#define REG_TILE_NBASELINE ((NSTATION/2+1)*(NSTATION/4))

#ifndef FIXED_POINT
// texture declaration for FP32 reads
static texture<float2, 1, hipReadModeElementType> tex1dfloat2;
static texture<float2, 2, hipReadModeElementType> tex2dfloat2;
#else
// texture declaration for 8-bit fixed point reads
static texture<char2, 1, hipReadModeNormalizedFloat> tex1dfloat2;
static texture<char2, 2, hipReadModeNormalizedFloat> tex2dfloat2;
#endif

// array holding indices for which matrix we are doing the output to at a given iteration
#if (NPULSAR > 0)
static __device__ __constant__ unsigned char tIndex[PIPE_LENGTH*NFREQUENCY];
#endif

#define checkCudaError() do {                           \
    hipError_t error = hipGetLastError();		\
    if (error != hipSuccess) {				\
      fprintf(stderr, "(CUDA) %s", hipGetErrorString(error));	\
      fprintf(stderr, " (" __FILE__ ":%d)\n", __LINE__);		\
      return XGPU_CUDA_ERROR;						\
    }							\
  } while (0)


//determine row and column from blockIdx.x
CUBE_DEVICE(static void, findPosition, unsigned int &Col, unsigned int &Row, unsigned int &blockX, unsigned int &blockY) {
  unsigned int k = blockIdx.x;
  blockY = -0.5f + sqrtf(0.25f + 2*k);
  blockX = k - (((blockY+1)*(blockY)) >> 1);
  Row = (blockY*TILE_HEIGHT + threadIdx.y);
  Col = (blockX*TILE_WIDTH + threadIdx.x);
}

__device__ static void operator+=( float4 &a, const float4 b ) {
 float4 t = a;
 t.x += b.x; t.y += b.y; t.z += b.z; t.w += b.w;
 a = t;
}

// device function to write out the matrix elements
CUBE_DEVICE(static void, write2x2, unsigned int &Col, unsigned int &Row, float4 *matrix_real, float4 *matrix_imag, 
	    float sum11XXreal, float sum11XXimag, float sum11XYreal, float sum11XYimag,
	    float sum11YXreal, float sum11YXimag, float sum11YYreal, float sum11YYimag,
	    float sum12XXreal, float sum12XXimag, float sum12XYreal, float sum12XYimag,
	    float sum12YXreal, float sum12YXimag, float sum12YYreal, float sum12YYimag,
	    float sum21XXreal, float sum21XXimag, float sum21XYreal, float sum21XYimag,
	    float sum21YXreal, float sum21YXimag, float sum21YYreal, float sum21YYimag,
	    float sum22XXreal, float sum22XXimag, float sum22XYreal, float sum22XYimag,
	    float sum22YXreal, float sum22YXimag, float sum22YYreal, float sum22YYimag) {
  
  int f=blockIdx.y;

#if (MATRIX_ORDER == REGISTER_TILE_TRIANGULAR_ORDER) // write out the register tiles separately
  matrix_real[f*4*REG_TILE_NBASELINE + REG_TILE_NBASELINE*0 + (Row*(Row+1)/2) + Col] += 
    make_float4(SCALE*sum11XXreal, SCALE*sum11XYreal, SCALE*sum11YXreal, SCALE*sum11YYreal);
  matrix_imag[f*4*REG_TILE_NBASELINE + REG_TILE_NBASELINE*0 + (Row*(Row+1)/2) + Col] += 
    make_float4(SCALE*sum11XXimag, SCALE*sum11XYimag, SCALE*sum11YXimag, SCALE*sum11YYimag);

  matrix_real[f*4*REG_TILE_NBASELINE + REG_TILE_NBASELINE*1 + (Row*(Row+1)/2) + Col] += 
    make_float4(SCALE*sum21XXreal, SCALE*sum21XYreal, SCALE*sum21YXreal, SCALE*sum21YYreal);
  matrix_imag[f*4*REG_TILE_NBASELINE + REG_TILE_NBASELINE*1 + (Row*(Row+1)/2) + Col] += 
    make_float4(SCALE*sum21XXimag, SCALE*sum21XYimag, SCALE*sum21YXimag, SCALE*sum21YYimag);

  matrix_real[f*4*REG_TILE_NBASELINE + REG_TILE_NBASELINE*3 + (Row*(Row+1)/2) + Col] += 
    make_float4(SCALE*sum22XXreal, SCALE*sum22XYreal, SCALE*sum22YXreal, SCALE*sum22YYreal);
  matrix_imag[f*4*REG_TILE_NBASELINE + REG_TILE_NBASELINE*3 + (Row*(Row+1)/2) + Col] += 
    make_float4(SCALE*sum22XXimag, SCALE*sum22XYimag, SCALE*sum22YXimag, SCALE*sum22YYimag);
  
  // Test if entire tile needs to be written or just 3 of 4 parts (exclude top-right)
  if (Col<Row) {
    matrix_real[f*4*REG_TILE_NBASELINE + REG_TILE_NBASELINE*2 + (Row*(Row+1)/2) + Col] += 
      make_float4(SCALE*sum12XXreal, SCALE*sum12XYreal, SCALE*sum12YXreal, SCALE*sum12YYreal);
    matrix_imag[f*4*REG_TILE_NBASELINE + REG_TILE_NBASELINE*2 + (Row*(Row+1)/2) + Col] += 
      make_float4(SCALE*sum12XXimag, SCALE*sum12XYimag, SCALE*sum12YXimag, SCALE*sum12YYimag);
  }
#elif (MATRIX_ORDER == REAL_IMAG_TRIANGULAR_ORDER) // write out the real and imaginary components separately
  Col*=2; Row*=2;
  matrix_real[f*NBASELINE + (Row*(Row+1)/2) + Col] += 
    make_float4(SCALE*sum11XXreal, SCALE*sum11XYreal, SCALE*sum11YXreal, SCALE*sum11YYreal);
  matrix_imag[f*NBASELINE + (Row*(Row+1)/2) + Col] += 
    make_float4(SCALE*sum11XXimag, SCALE*sum11XYimag, SCALE*sum11YXimag, SCALE*sum11YYimag);

  matrix_real[f*NBASELINE + ((Row+1)*(Row+2)/2) + Col] += 
    make_float4(SCALE*sum21XXreal, SCALE*sum21XYreal, SCALE*sum21YXreal, SCALE*sum21YYreal);
  matrix_imag[f*NBASELINE + ((Row+1)*(Row+2)/2) + Col] += 
    make_float4(SCALE*sum21XXimag, SCALE*sum21XYimag, SCALE*sum21YXimag, SCALE*sum21YYimag);

  matrix_real[f*NBASELINE + ((Row+1)*(Row+2)/2) + (Col+1)] += 
    make_float4(SCALE*sum22XXreal, SCALE*sum22XYreal, SCALE*sum22YXreal, SCALE*sum22YYreal);
  matrix_imag[f*NBASELINE + ((Row+1)*(Row+2)/2) + (Col+1)] += 
    make_float4(SCALE*sum22XXimag, SCALE*sum22XYimag, SCALE*sum22YXimag, SCALE*sum22YYimag);
  
  // Test if entire tile needs to be written or just 3 of 4 parts (exclude top-right)
  if (Col<Row) {
    matrix_real[f*NBASELINE + (Row*(Row+1)/2) + (Col+1)] += 
      make_float4(SCALE*sum12XXreal, SCALE*sum12XYreal, SCALE*sum12YXreal, SCALE*sum12YYreal);
    matrix_imag[f*NBASELINE + (Row*(Row+1)/2) + (Col+1)] += 
      make_float4(SCALE*sum12XXimag, SCALE*sum12XYimag, SCALE*sum12YXimag, SCALE*sum12YYimag);
  }
#else  // standard triangular packed order
  Col*=2; Row*=2;
  matrix_real[(f*NBASELINE + (Row*(Row+1)/2) + Col)*NPOL + 0] += 
    make_float4(SCALE*sum11XXreal, SCALE*sum11XXimag, SCALE*sum11XYreal, SCALE*sum11XYimag);
  matrix_real[(f*NBASELINE + (Row*(Row+1)/2) + Col)*NPOL + 1] += 
    make_float4(SCALE*sum11YXreal, SCALE*sum11YXimag, SCALE*sum11YYreal, SCALE*sum11YYimag);
  matrix_real[(f*NBASELINE + ((Row+1)*(Row+2)/2) + Col)*NPOL + 0] += 
    make_float4(SCALE*sum21XXreal, SCALE*sum21XXimag, SCALE*sum21XYreal, SCALE*sum21XYimag);
  matrix_real[(f*NBASELINE + ((Row+1)*(Row+2)/2) + Col)*NPOL + 1] += 
    make_float4(SCALE*sum21YXreal, SCALE*sum21YXimag, SCALE*sum21YYreal, SCALE*sum21YYimag);
  matrix_real[(f*NBASELINE + ((Row+1)*(Row+2)/2) + (Col+1))*NPOL + 0] += 
    make_float4(SCALE*sum22XXreal, SCALE*sum22XXimag, SCALE*sum22XYreal, SCALE*sum22XYimag);
  matrix_real[(f*NBASELINE + ((Row+1)*(Row+2)/2) + (Col+1))*NPOL + 1] += 
    make_float4(SCALE*sum22YXreal, SCALE*sum22YXimag, SCALE*sum22YYreal, SCALE*sum22YYimag);
  
  // Test if entire tile needs to be written or just 3 of 4 parts (exclude top-right)
  if (Col<Row) {
    matrix_real[(f*NBASELINE + (Row*(Row+1)/2) + (Col+1))*NPOL + 0] += 
      make_float4(SCALE*sum12XXreal, SCALE*sum12XXimag, SCALE*sum12XYreal, SCALE*sum12XYimag);
    matrix_real[(f*NBASELINE + (Row*(Row+1)/2) + (Col+1))*NPOL + 1] += 
      make_float4(SCALE*sum12YXreal, SCALE*sum12YXimag, SCALE*sum12YYreal, SCALE*sum12YYimag);
  }
#endif

}

// Define TEXTURE_DIM as 1 to use 1D texture (more accurate, costs 1 mult per LOAD)
// Define TEXTURE_DIM as 2 to use 2D texture (less accurate, saves 1 mult per LOAD)
#ifndef TEXTURE_DIM
#define TEXTURE_DIM 1
#endif

#if TEXTURE_DIM == 1
// Read in column in first warp as float2, row in second warp (still true for 1D?)
#define LOAD(s, t)							\
  {float2 temp = tex1Dfetch(tex1dfloat2, array_index + (t)*NFREQUENCY*Nstation*NPOL);			\
    CUBE_ADD_BYTES(sizeof(ComplexInput));				\
    *(input##s##_p) = temp.x;						\
    *(input##s##_p + 4*TILE_WIDTH) = temp.y;}

#elif TEXTURE_DIM == 2
// Read in column in first warp as float2, row in second warp
#define LOAD(s, t)							\
  {float2 temp = tex2D(tex2dfloat2, array_index, t);			\
    CUBE_ADD_BYTES(sizeof(ComplexInput));				\
    *(input##s##_p) = temp.x;						\
    *(input##s##_p + 4*TILE_WIDTH) = temp.y;}

#else
#error TEXTURE_DIM must be 1 or 2
#endif

// read in shared data as individual floats to avoid bank conflicts

#define TWO_BY_TWO_COMPUTE(s)						\
  {float col1Xreal = input[s][4*tx];					\
  float col1Ximag = input[s][4*tx + 4*TILE_WIDTH];			\
  float col1Yreal = input[s][4*tx + 1];					\
  float col1Yimag = input[s][4*tx + 1 + 4*TILE_WIDTH];			\
  float col2Xreal = input[s][4*tx + 2];					\
  float col2Ximag = input[s][4*tx + 2 + 4*TILE_WIDTH];			\
  float col2Yreal = input[s][4*tx + 3];					\
  float col2Yimag = input[s][4*tx + 3 + 4*TILE_WIDTH];			\
  float row1Xreal = input[s][4*ty + 8*TILE_WIDTH];			\
  float row1Ximag = input[s][4*ty + 4*TILE_HEIGHT + 8*TILE_WIDTH];	\
  float row1Yreal = input[s][4*ty + 1 + 8*TILE_WIDTH];			\
  float row1Yimag = input[s][4*ty + 1 + 4*TILE_HEIGHT + 8*TILE_WIDTH];	\
  float row2Xreal = input[s][4*ty + 2 + 8*TILE_WIDTH];			\
  float row2Ximag = input[s][4*ty + 2 + 4*TILE_HEIGHT + 8*TILE_WIDTH];	\
  float row2Yreal = input[s][4*ty + 3 + 8*TILE_WIDTH];			\
  float row2Yimag = input[s][4*ty + 3 + 4*TILE_HEIGHT + 8*TILE_WIDTH];	\
  sum11XXreal += row1Xreal * col1Xreal;					\
  sum11XXreal += row1Ximag * col1Ximag;					\
  sum11XXimag += row1Ximag * col1Xreal;					\
  sum11XXimag -= row1Xreal * col1Ximag;					\
  sum11XYreal += row1Xreal * col1Yreal;					\
  sum11XYreal += row1Ximag * col1Yimag;					\
  sum11XYimag += row1Ximag * col1Yreal;					\
  sum11XYimag -= row1Xreal * col1Yimag;					\
  sum11YXreal += row1Yreal * col1Xreal;					\
  sum11YXreal += row1Yimag * col1Ximag;					\
  sum11YXimag += row1Yimag * col1Xreal;					\
  sum11YXimag -= row1Yreal * col1Ximag;					\
  sum11YYreal += row1Yreal * col1Yreal;					\
  sum11YYreal += row1Yimag * col1Yimag;					\
  sum11YYimag += row1Yimag * col1Yreal;					\
  sum11YYimag -= row1Yreal * col1Yimag;					\
  sum12XXreal += row1Xreal * col2Xreal;					\
  sum12XXreal += row1Ximag * col2Ximag;					\
  sum12XXimag += row1Ximag * col2Xreal;					\
  sum12XXimag -= row1Xreal * col2Ximag;					\
  sum12XYreal += row1Xreal * col2Yreal;					\
  sum12XYreal += row1Ximag * col2Yimag;					\
  sum12XYimag += row1Ximag * col2Yreal;					\
  sum12XYimag -= row1Xreal * col2Yimag;					\
  sum12YXreal += row1Yreal * col2Xreal;					\
  sum12YXreal += row1Yimag * col2Ximag;					\
  sum12YXimag += row1Yimag * col2Xreal;					\
  sum12YXimag -= row1Yreal * col2Ximag;					\
  sum12YYreal += row1Yreal * col2Yreal;					\
  sum12YYreal += row1Yimag * col2Yimag;					\
  sum12YYimag += row1Yimag * col2Yreal;					\
  sum12YYimag -= row1Yreal * col2Yimag;					\
  sum21XXreal += row2Xreal * col1Xreal;					\
  sum21XXreal += row2Ximag * col1Ximag;					\
  sum21XXimag += row2Ximag * col1Xreal;					\
  sum21XXimag -= row2Xreal * col1Ximag;					\
  sum21XYreal += row2Xreal * col1Yreal;					\
  sum21XYreal += row2Ximag * col1Yimag;					\
  sum21XYimag += row2Ximag * col1Yreal;					\
  sum21XYimag -= row2Xreal * col1Yimag;					\
  sum21YXreal += row2Yreal * col1Xreal;					\
  sum21YXreal += row2Yimag * col1Ximag;					\
  sum21YXimag += row2Yimag * col1Xreal;					\
  sum21YXimag -= row2Yreal * col1Ximag;					\
  sum21YYreal += row2Yreal * col1Yreal;					\
  sum21YYreal += row2Yimag * col1Yimag;					\
  sum21YYimag += row2Yimag * col1Yreal;					\
  sum21YYimag -= row2Yreal * col1Yimag;					\
  sum22XXreal += row2Xreal * col2Xreal;					\
  sum22XXreal += row2Ximag * col2Ximag;					\
  sum22XXimag += row2Ximag * col2Xreal;					\
  sum22XXimag -= row2Xreal * col2Ximag;					\
  sum22XYreal += row2Xreal * col2Yreal;					\
  sum22XYreal += row2Ximag * col2Yimag;					\
  sum22XYimag += row2Ximag * col2Yreal;					\
  sum22XYimag -= row2Xreal * col2Yimag;					\
  sum22YXreal += row2Yreal * col2Xreal;					\
  sum22YXreal += row2Yimag * col2Ximag;					\
  sum22YXimag += row2Yimag * col2Xreal;					\
  sum22YXimag -= row2Yreal * col2Ximag;					\
  sum22YYreal += row2Yreal * col2Yreal;					\
  sum22YYreal += row2Yimag * col2Yimag;					\
  sum22YYimag += row2Yimag * col2Yreal;					\
  sum22YYimag -= row2Yreal * col2Yimag;}

CUBE_KERNEL(static shared2x2float2, float4 *matrix_real, float4 *matrix_imag, const int Nstation, const int write)
{
  CUBE_START;

  //get local thread ID
  unsigned int ty = threadIdx.y;
  unsigned int tx = threadIdx.x;
  unsigned int tid = ty*TILE_WIDTH + tx;

  //set frequency number from blockIdx.y
  unsigned int f = blockIdx.y;

  unsigned int Row, Col, blockX, blockY;
  CUBE_DEVICE_CALL(findPosition, Col, Row, blockX, blockY);

  //declare shared memory for input coalescing
  __shared__ float input[2][16*TILE_WIDTH]; // 4* for float4, 2* for 2x2 tile size

  //instantiate sum variables
  float sum11XXreal = 0.0, sum11XXimag = 0.0;
  float sum11XYreal = 0.0, sum11XYimag = 0.0;
  float sum11YXreal = 0.0, sum11YXimag = 0.0;
  float sum11YYreal = 0.0, sum11YYimag = 0.0;
  float sum12XXreal = 0.0, sum12XXimag = 0.0;
  float sum12XYreal = 0.0, sum12XYimag = 0.0;
  float sum12YXreal = 0.0, sum12YXimag = 0.0;
  float sum12YYreal = 0.0, sum12YYimag = 0.0;
  float sum21XXreal = 0.0, sum21XXimag = 0.0;
  float sum21XYreal = 0.0, sum21XYimag = 0.0;
  float sum21YXreal = 0.0, sum21YXimag = 0.0;
  float sum21YYreal = 0.0, sum21YYimag = 0.0;
  float sum22XXreal = 0.0, sum22XXimag = 0.0;
  float sum22XYreal = 0.0, sum22XYimag = 0.0;
  float sum22YXreal = 0.0, sum22YXimag = 0.0;
  float sum22YYreal = 0.0, sum22YYimag = 0.0;

  float *input0_p = input[0] + tid;
  float *input1_p = input[1] + tid;
  unsigned int array_index = f*Nstation*NPOL + tid;
  //float array_index = f*Nstation*NPOL + tid;
  if (tid < 4*TILE_WIDTH) {
    array_index += 2*blockX*TILE_WIDTH*NPOL;
  } else {
    array_index += 2*blockY*TILE_WIDTH*NPOL - 4*TILE_HEIGHT;    
    input0_p += 4*TILE_WIDTH;
    input1_p += 4*TILE_WIDTH;
  }


  LOAD(0, 0);

#pragma unroll 2
  for(unsigned int t=0; t<NTIME_PIPE-2; t+=2){
    //for(float t=0.0f; t<(float)NTIME_PIPE-2.0f; /*t+=2.0f*/){

    __syncthreads();

    TWO_BY_TWO_COMPUTE(0);

    //t += 1.0f;
    //LOAD(1, t);    
    LOAD(1, t+1);

    __syncthreads();

    TWO_BY_TWO_COMPUTE(1);

    //t += 1.0f;
    //LOAD(0, t);
    LOAD(0, t+2);
  } 

  __syncthreads();  
  TWO_BY_TWO_COMPUTE(0);

  LOAD(1, NTIME_PIPE-1);

  __syncthreads();

  if (Col > Row) return; // writes seem faster when this is pulled up here
  TWO_BY_TWO_COMPUTE(1);

#ifdef WRITE_OPTION
  if (write) {
#endif
    CUBE_DEVICE_CALL(write2x2, Col, Row, matrix_real, matrix_imag,
		     sum11XXreal, sum11XXimag, sum11XYreal, sum11XYimag, 
		     sum11YXreal, sum11YXimag, sum11YYreal, sum11YYimag, 
		     sum12XXreal, sum12XXimag, sum12XYreal, sum12XYimag, 
		     sum12YXreal, sum12YXimag, sum12YYreal, sum12YYimag, 
		     sum21XXreal, sum21XXimag, sum21XYreal, sum21XYimag, 
		     sum21YXreal, sum21YXimag, sum21YYreal, sum21YYimag, 
		     sum22XXreal, sum22XXimag, sum22XYreal, sum22XYimag, 
		     sum22YXreal, sum22YXimag, sum22YYreal, sum22YYimag);

    CUBE_ADD_BYTES(Col < Row ? 256 : 192); // need load and save
#ifdef WRITE_OPTION
  }
#endif

  CUBE_ADD_FLOPS(NTIME_PIPE*(Col < Row ? 128 : 96));

  CUBE_END;
}

#undef LOAD
#undef TWO_BY_TWO_COMPUTE

static XGPUInfo compiletime_info = {
  npol:          NPOL,
  nstation:      NSTATION,
  nbaseline:     NBASELINE,
  nfrequency:    NFREQUENCY,
  ntime:         NTIME,
  ntimepipe:     NTIME_PIPE,
#ifdef FIXED_POINT
  input_type:    XGPU_INT8,
#else
  input_type:    XGPU_FLOAT32,
#endif
  vecLength:     NFREQUENCY * NTIME * NSTATION * NPOL,
  vecLengthPipe: NFREQUENCY * NTIME_PIPE * NSTATION * NPOL,
#if (MATRIX_ORDER == REGISTER_TILE_TRIANGULAR_ORDER)
  matLength:     NFREQUENCY * ((NSTATION/2+1)*(NSTATION/4)*NPOL*NPOL*4) * (NPULSAR + 1),
#else
  // Matrix length is same for REGISTER_TILE_TRIANGULAR_ORDER and TRIANGULAR_ORDER
  matLength:     NFREQUENCY * ((NSTATION+1)*(NSTATION/2)*NPOL*NPOL) * (NPULSAR + 1),
#endif
  matrix_order:  MATRIX_ORDER
};

// This stringification trick is from "info cpp"
#define STRINGIFY1(s) #s
#define STRINGIFY(s) STRINGIFY1(s)
static const char xgpu_version[] = STRINGIFY(XGPU_VERSION);

const char * xgpuVersionString()
{
  return xgpu_version;
}

// Populate XGPUInfo structure with compile-time parameters.
void xgpuInfo(XGPUInfo *pcxs)
{
  pcxs->npol           = compiletime_info.npol;
  pcxs->nstation       = compiletime_info.nstation;
  pcxs->nbaseline      = compiletime_info.nbaseline;
  pcxs->nfrequency     = compiletime_info.nfrequency;
  pcxs->ntime          = compiletime_info.ntime;
  pcxs->ntimepipe      = compiletime_info.ntimepipe;
  pcxs->input_type     = compiletime_info.input_type;
  pcxs->vecLength      = compiletime_info.vecLength;
  pcxs->vecLengthPipe  = compiletime_info.vecLengthPipe;
  pcxs->matLength      = compiletime_info.matLength;
  pcxs->matrix_order   = compiletime_info.matrix_order;
}

// Initialize the XGPU.
int xgpuInit(XGPUContext *context)
{

  CUBE_INIT();

  // Allocate internal context
  XGPUInternalContext *internal = (XGPUInternalContext *)malloc(sizeof(XGPUInternalContext));
  if(!internal) {
    // Uh-oh!
    return XGPU_OUT_OF_MEMORY;
  }
  context->internal = internal;

  long long unsigned int vecLength = compiletime_info.vecLength;
  long long unsigned int vecLengthPipe = compiletime_info.vecLengthPipe;
  long long unsigned int matLength = compiletime_info.matLength;

  //assign the device
  const int device = 0;
  hipSetDevice(device); // TODO Put device number in XGPU(Internal?)Context
  checkCudaError();

  if(context->array_h) {
    // Register caller-allocated host memory with CUDA.
    // This requires that the caller allocated the memory properly vis-a-vis
    // the requirements of hipHostRegister!
    hipHostRegister(context->array_h, compiletime_info.vecLength*sizeof(ComplexInput), 0);
    internal->unregister_array_h = context->array_h;
    internal->free_array_h = NULL;
  } else {
    // allocate host memory
    hipHostMalloc(&(context->array_h), vecLength*sizeof(ComplexInput));
    internal->free_array_h = context->array_h;
    internal->unregister_array_h = NULL;
    checkCudaError();
  }

  if(context->matrix_h) {
    // Register caller-allocated host memory with CUDA.
    // This requires that the caller allocated the memory properly vis-a-vis
    // the requirements of hipHostRegister!
    hipHostRegister(context->matrix_h, compiletime_info.vecLength*sizeof(ComplexInput), 0);
    internal->unregister_matrix_h = context->matrix_h;
    internal->free_matrix_h = NULL;
  } else {
    // allocate host memory
    hipHostMalloc(&(context->matrix_h), matLength*sizeof(Complex));
    internal->free_matrix_h = context->matrix_h;
    internal->unregister_matrix_h = NULL;
    checkCudaError();
  }

  //allocate memory on device
  hipMalloc((void **) &(internal->array_d[0]), vecLengthPipe*sizeof(ComplexInput));
  hipMalloc((void **) &(internal->array_d[1]), vecLengthPipe*sizeof(ComplexInput));
  hipMalloc((void **) &(internal->matrix_d), matLength*sizeof(Complex));
  checkCudaError();
  
  //clear out any previous values
  hipMemset(internal->array_d[0], '\0', vecLengthPipe*sizeof(ComplexInput));
  hipMemset(internal->array_d[1], '\0', vecLengthPipe*sizeof(ComplexInput));
  hipMemset(internal->matrix_d, '\0', matLength*sizeof(Complex));
  checkCudaError();

  // create the streams
  internal->streams = (hipStream_t*) malloc(2*sizeof(hipStream_t));
  for(int i=0; i<2; i++) hipStreamCreate(&(internal->streams[i]));
  checkCudaError();

  internal->channelDesc = hipCreateChannelDesc<COMPLEX_INPUT>();

#if NPULSAR > 0
  unsigned char timeIndex[PIPE_LENGTH*NFREQUENCY];
  for (int tf=0; tf<PIPE_LENGTH*NFREQUENCY; tf++) timeIndex[tf] = 0;
  hipMemcpyToSymbol(HIP_SYMBOL(tIndex), timeIndex, PIPE_LENGTH*NFREQUENCY*sizeof(unsigned char), hipMemcpyHostToDevice);

  checkCudaError();

  // check symbols are copied over
  unsigned char timeIndex2[PIPE_LENGTH*NFREQUENCY];
  hipMemcpyFromSymbol(timeIndex2[t], HIP_SYMBOL(tIndex[t]), PIPE_LENGTH*NFREQUENCY*sizeof(unsigned char), hipMemcpyDeviceToHost);  
  for (int tf=0; tf<PIPE_LENGTH*NFREQUENCY; tf++) {
    for (int f=0; f<NFREQUENCY; f++) 
      if (timeIndex[t][f] != timeIndex2[t][f]) 
	fprintf(stderr, "Index copy failed: t = %d, f = %d, original = %d, copy = %d\n", 
	       t, f, timeIndex[t][f], timeIndex2[t][f]);
  }
#endif

  // check whether texture dimensions are ok
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, device);

#if TEXTURE_DIM == 2
  if((NFREQUENCY * NSTATION * NPOL > deviceProp.maxTexture2D[0]) ||
     (NTIME_PIPE > deviceProp.maxTexture2D[1])) {
    return XGPU_INSUFFICIENT_TEXTURE_MEMORY;
  }
#elif TEXTURE_DIM == 1
#if 0
  // Surprisingly, this appears not to be a problem with 1D textures.  On a
  // GeForce GTX 580 (i.e. Fermi device), deviceQuery returns 65536 as
  // maxTexture1D, yet the default sizes use 10 * 256 * 2 * 100 * 2 == 1024000
  // bytes of 1D texture without any problems.  Perhaps the value of
  // maxTexture1D returned by hipGetDeviceProperties is wrong?
  if (NFREQUENCY * NSTATION * NPOL * NTIME_PIPE > deviceProp.maxTexture1D) {
    return XGPU_INSUFFICIENT_TEXTURE_MEMORY;
  }
#endif
#endif 

  return XGPU_OK;
}

// Reinitialize the XGPU host buffers.
int xgpuReinit(XGPUContext *context)
{
  XGPUInternalContext *internal = (XGPUInternalContext *)context->internal;
  if(!internal) {
    return XGPU_NOT_INITIALIZED;
  }

  if(context->array_h) {
    if(internal->free_array_h) {
      hipHostFree(internal->free_array_h);
    }
    if(internal->unregister_array_h) {
      hipHostUnregister(internal->unregister_array_h);
    }
    // Register caller-allocated host memory with CUDA.
    // This requires that the caller allocated the memory properly vis-a-vis
    // the requirements of hipHostRegister!
    hipHostRegister(context->array_h, compiletime_info.vecLength*sizeof(ComplexInput), 0);
    internal->unregister_array_h = context->array_h;
    internal->free_array_h = NULL;
  }

  if(context->matrix_h) {
    if(internal->free_matrix_h) {
      hipHostFree(internal->free_matrix_h);
    }
    if(internal->unregister_matrix_h) {
      hipHostUnregister(internal->unregister_matrix_h);
    }
    // Register caller-allocated host memory with CUDA.
    // This requires that the caller allocated the memory properly vis-a-vis
    // the requirements of hipHostRegister!
    hipHostRegister(context->matrix_h, compiletime_info.vecLength*sizeof(ComplexInput), 0);
    internal->unregister_matrix_h = context->matrix_h;
    internal->free_matrix_h = NULL;
  }

  return XGPU_OK;
}

// Free up the memory on the host and device
void xgpuFree(XGPUContext *context)
{
  XGPUInternalContext *internal = (XGPUInternalContext *)context->internal;

  if(internal) {
    for(int i=0; i<2; i++)
      hipStreamDestroy(internal->streams[i]);

    if(internal->free_array_h) {
      hipHostFree(internal->free_array_h);
      context->array_h = NULL;
    }
    if(internal->unregister_array_h) {
      hipHostUnregister(internal->unregister_array_h);
      context->array_h = NULL;
    }
    if(internal->free_matrix_h) {
      hipHostFree(internal->free_matrix_h);
      context->matrix_h = NULL;
    }
    if(internal->unregister_matrix_h) {
      hipHostUnregister(internal->unregister_matrix_h);
      context->matrix_h = NULL;
    }

    hipFree(internal->array_d[1]);
    hipFree(internal->array_d[0]);
    hipFree(internal->matrix_d);

    free(internal);
    context->internal = NULL;
  }

  CUBE_WRITE();
}

int xgpuCudaXengine(XGPUContext *context)
{
  XGPUInternalContext *internal = (XGPUInternalContext *)context->internal;
  if(!internal) {
    return XGPU_NOT_INITIALIZED;
  }

  ComplexInput **array_d = internal->array_d;
  hipStream_t *streams = internal->streams;
  hipChannelFormatDesc channelDesc = internal->channelDesc;

  // set pointers to the real and imaginary components of the device matrix
  float4 *matrix_real_d = (float4 *)(internal->matrix_d);
  float4 *matrix_imag_d = (float4 *)(internal->matrix_d + compiletime_info.matLength/2);

  int Nblock = compiletime_info.nstation/min(TILE_HEIGHT,TILE_WIDTH);
  ComplexInput *array_load;
  ComplexInput *array_compute; 

  dim3 dimBlock(TILE_WIDTH,TILE_HEIGHT,1);
  //allocated exactly as many thread blocks as are needed
  dim3 dimGrid(((Nblock/2+1)*(Nblock/2))/2, compiletime_info.nfrequency);

  // Create events used to record the completion of the device-host transfer and kernels
  hipEvent_t copyCompletion[2], kernelCompletion[2];
  for (int i=0; i<2; i++) {
    hipEventCreate(&kernelCompletion[i]);
    hipEventCreate(&copyCompletion[i]);
  }
  checkCudaError();

  CUBE_ASYNC_START(ENTIRE_PIPELINE);

  // Need to fill pipeline before loop
  long long unsigned int vecLengthPipe = compiletime_info.vecLengthPipe;
  ComplexInput *array_hp = &context->array_h[0*vecLengthPipe];
  CUBE_ASYNC_COPY_CALL(array_d[0], array_hp, vecLengthPipe*sizeof(ComplexInput), hipMemcpyHostToDevice, streams[0]);
  hipEventRecord(copyCompletion[0], streams[0]); // record the completion of the h2d transfer
  checkCudaError();

  CUBE_ASYNC_START(PIPELINE_LOOP);

#ifdef POWER_LOOP
  for (int q=0; ; q++) 
#endif
  for (int p=1; p<PIPE_LENGTH; p++) {
    array_compute = array_d[(p+1)%2];
    array_load = array_d[p%2];

    // Kernel Calculation
#if TEXTURE_DIM == 2
    hipBindTexture2D(0, tex2dfloat2, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL, NTIME_PIPE, 
		      NFREQUENCY*Nstation*NPOL*sizeof(ComplexInput));
#else
    hipBindTexture(0, tex1dfloat2, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL*NTIME_PIPE*sizeof(ComplexInput));
#endif
    hipStreamWaitEvent(streams[1], copyCompletion[(p+1)%2], 0); // only start the kernel once the h2d transfer is complete
    CUBE_ASYNC_KERNEL_CALL(shared2x2float2, dimGrid, dimBlock, 0, streams[1], 
			   matrix_real_d, matrix_imag_d, NSTATION, writeMatrix);
    hipEventRecord(kernelCompletion[(p+1)%2], streams[1]); // record the completion of the h2d transfer
    checkCudaError();

    // Download input data
    ComplexInput *array_hp = &context->array_h[p*vecLengthPipe];
    hipStreamWaitEvent(streams[0], kernelCompletion[p%2], 0); // only start the transfer once the kernel has completed
    CUBE_ASYNC_COPY_CALL(array_load, array_hp, vecLengthPipe*sizeof(ComplexInput), hipMemcpyHostToDevice, streams[0]);
    hipEventRecord(copyCompletion[p%2], streams[0]); // record the completion of the h2d transfer
    checkCudaError();
  }

  CUBE_ASYNC_END(PIPELINE_LOOP);

  array_compute = array_d[(PIPE_LENGTH+1)%2];
  // Final kernel calculation
#if TEXTURE_DIM == 2
  hipBindTexture2D(0, tex2dfloat2, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL, NTIME_PIPE, 
		    NFREQUENCY*Nstation*NPOL*sizeof(ComplexInput));
#else
    hipBindTexture(0, tex1dfloat2, array_compute, channelDesc, NFREQUENCY*NSTATION*NPOL*NTIME_PIPE*sizeof(ComplexInput));
#endif
  hipStreamWaitEvent(streams[1], copyCompletion[(PIPE_LENGTH+1)%2], 0);
  CUBE_ASYNC_KERNEL_CALL(shared2x2float2, dimGrid, dimBlock, 0, streams[1], matrix_real_d, matrix_imag_d,
			 NSTATION, writeMatrix);
  checkCudaError();

  //copy the data back, employing a similar strategy as above
  CUBE_COPY_CALL(context->matrix_h, internal->matrix_d, compiletime_info.matLength*sizeof(Complex), hipMemcpyDeviceToHost);
  checkCudaError();

  CUBE_ASYNC_END(ENTIRE_PIPELINE);

  for (int i=0; i<2; i++) {
    hipEventDestroy(copyCompletion[i]);
    hipEventDestroy(kernelCompletion[i]);
  }
  checkCudaError();

  return XGPU_OK;
}
